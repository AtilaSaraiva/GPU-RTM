#include "hip/hip_runtime.h"
/*
Hello world of wave propagation in CUDA. FDTD acoustic wave propagation in homogeneous medium. Second order accurate in time and eigth in space.

Oleg Ovcharenko
Vladimir Kazei, 2019

oleg.ovcharenko@kaust.edu.sa
vladimir.kazei@kaust.edu.sa
*/

#include <rsf.hh>
#include <iostream>
#include <string>
#include "stdio.h"
#include "math.h"
#include "stdlib.h"
#include "string.h"

#include "btree.cuh"

using namespace std;

/*
Add this to c_cpp_properties.json if linting isn't working for CUDA libraries
"includePath": [
                "/usr/local/cuda-10.0/targets/x86_64-linux/include",
                "${workspaceFolder}/**"
            ],
*/

//typedef struct{
    //int nShots;
    //int srcPosX;
    //int srcPosY;
    //int firstReceptorPos;
    //int nReceptors;
    //int lastReceptorPos;
    //int incShots;
    //int incRec;
    //int modelNx;
    //int modelNy;
    //int modelNxBorder;
    //int modelNyBorder;
    //float modelDx;
    //float modelDy;
    //int taperBorder;
    //// Auxiliaries
    //size_t nxy;
    //size_t nbxy;
    //size_t nbytes;
//} geometry;

//typedef struct{
    //float *velField;
    //float *extVelField;
    //float *firstLayerVelField;
    //float *reflecitivy;
    //float maxVel;
//} velocity;

//typedef struct{
    //float timeStep;
    //int timeSamplesNt;
    //float *seismogram;
//} seismicData;

//#include "cuwaveprop2d.cu"

//using namespace std;

//void dummyVelField(int nxb, int nyb, int nb, float *h_vpe, float *h_dvpe)
//{
    //for (int i = 0; i < nyb; i++){
        //for (int j = 0; j < nxb; j++){
            //h_dvpe[j * nyb + i]  = h_vpe[j * nyb + nb];
        //}
    //}
//}

//void expand(int nb, int nyb, int nxb, int nz, int nx, float *a, float *b)
//[>< expand domain of 'a' to 'b':  a, size=nz*nx; b, size=nyb*nxb;  ><]
//{
    //int iz,ix;
    //for     (ix=0;ix<nx;ix++) {
        //for (iz=0;iz<nz;iz++) {
            //b[(nb+ix)*nyb+(nb+iz)] = a[ix*nz+iz];
        //}
    //}
    //for     (ix=0; ix<nxb; ix++) {
        //for (iz=0; iz<nb; iz++)         b[ix*nyb+iz] = b[ix*nyb+nb];//top
        //for (iz=nz+nb; iz<nyb; iz++) b[ix*nyb+iz] = b[ix*nyb+nb+nz-1];//bottom
    //}
    //for (iz=0; iz<nyb; iz++){
        //for(ix=0; ix<nb; ix++)  b[ix*nyb+iz] = b[nb*nyb+iz];//left
        //for(ix=nb+nx; ix<nxb; ix++)     b[ix*nyb+iz] = b[(nb+nx-1)*nyb+iz];//right
    //}
//}

//void abc_coef (int nb, float *abc)
//{
    //for(int i=0; i<nb; i++){
        //abc[i] = exp (-pow(0.002 * (nb - i + 1),2.0));
    //}
//}

//void taper (int nx, int ny, int nb, float *abc, float *campo)
//{
    //int nxb = nx + 2 * nb;
    //int nyb = ny + 2 * nb;
    //for(int j=0; j<nxb; j++){
        //for(int i=0; i<nb; i++){
            //campo[j * nyb + i] *= abc[i];
            //campo[j * nyb + (nb + ny + i)] *= abc[nb - i - 1];
        //}
    //}
    //for(int i=0; i<nyb; i++){
        //for(int j=0; j<nb; j++){
            //campo[j * nyb + i] *= abc[j];
            //campo[(nb + nx + j) * nyb + i] *= abc[nb - j - 1];
        //}
    //}
//}

//sf_file createFile3D (const char *name, int dimensions[3], float spacings[3], int origins[3])
//{
    //sf_file Fdata = NULL;
    //Fdata = sf_output(name);
    //char key_n[6],key_d[6],key_o[6];
    //for (int i = 0; i < 3; i++){
        //sprintf(key_n,"n%i",i+1);
        //sprintf(key_d,"d%i",i+1);
        //sprintf(key_o,"o%i",i+1);
        //sf_putint(Fdata,key_n,dimensions[i]);
        //sf_putint(Fdata,key_d,spacings[i]);
        //sf_putint(Fdata,key_o,origins[i]);
    //}

    //return Fdata;
//}

//geometry getParameters(sf_file FvelModel, sf_file Fshots)
//{
    //geometry param;
    //sf_histint(Fshots,"n2",&param.nReceptors);
    //sf_histint(Fshots,"sybeg",&param.srcPosY);
    //sf_histint(Fshots,"sxbeg",&param.srcPosX);
    //sf_histint(Fshots,"gxbeg",&param.firstReceptorPos);
    //sf_histint(Fshots,"n3",&param.nShots);
    //sf_histint(Fshots,"incShots",&param.incShots);
    //sf_histint(Fshots,"incRec",&param.incRec);
    //sf_histint(FvelModel, "n1",&param.modelNy);
    //sf_histint(FvelModel, "n2", &param.modelNx);
    //sf_histfloat(FvelModel, "d1",&param.modelDy);
    //sf_histfloat(FvelModel, "d2", &param.modelDx);
    //param.lastReceptorPos = param.firstReceptorPos + param.nReceptors;
    //param.taperBorder = 0.3 * param.modelNx;
    //param.nxy = param.modelNx * param.modelNy;
    //param.modelNxBorder = param.modelNx + 2 * param.taperBorder;
    //param.modelNyBorder = param.modelNy + 2 * param.taperBorder;
    //param.nbxy = param.modelNxBorder * param.modelNyBorder;
    //param.nbytes = param.nbxy * sizeof(float); // bytes to store modelNxBorder * modelNyBorder
    //return param;
//}

//velocity getVelFields(sf_file FvelModel, geometry param)
//{
    //velocity h_model;

    //h_model.velField = new float[param.nxy];
    //sf_floatread(h_model.velField, param.nxy, FvelModel);

    //h_model.extVelField = new float[param.nbxy];
    //memset(h_model.extVelField,0,param.nbytes);
    //expand(param.taperBorder, param.modelNyBorder, param.modelNxBorder, param.modelNy, param.modelNx, h_model.velField, h_model.extVelField);

    //h_model.maxVel = h_model.velField[0];
    //for(int i=1; i < param.nxy; i++){
        //if(h_model.velField[i] > h_model.maxVel){
            //h_model.maxVel = h_model.velField[i];
        //}
    //}

    //h_model.firstLayerVelField = new float[param.nbxy];
    //dummyVelField(param.modelNxBorder, param.modelNyBorder, param.taperBorder, h_model.extVelField, h_model.firstLayerVelField);

    ////printf("MODEL:\n");
    ////printf("\t%i x %i\t:param.modelNy x param.modelNx\n", param.modelNy, param.modelNx);
    ////printf("\t%f\t:param.modelDx\n", param.modelDx);
    ////printf("\t%f\t:h_model.velField[0]\n", h_model.velField[0]);
    //return h_model;
//}

//float* tapermask(geometry param)
//{
    //float *h_abc = new float[param.taperBorder];
    //float *h_tapermask = new float[param.nbxy];
    //for(int i=0; i < param.nbxy; i++){
        //h_tapermask[i] = 1;
    //}
    //abc_coef(param.taperBorder, h_abc);
    //taper(param.modelNx, param.modelNy, param.taperBorder, h_abc, h_tapermask);
    //delete[] h_abc;
    //return h_tapermask;
//}

//seismicData allocHostSeisData(geometry param, sf_file Fshots)
//{
    //seismicData h_seisData;
    //sf_histfloat(Fshots,"d1",&h_seisData.timeStep);
    //sf_histint(Fshots,"n1",&h_seisData.timeSamplesNt);
    //h_seisData.seismogram = new float[param.nShots * param.nReceptors * h_seisData.timeSamplesNt];
    //sf_floatread(h_seisData.seismogram, param.nShots * param.nReceptors * h_seisData.timeSamplesNt, Fshots);
    //return h_seisData;
//}


//float* fillSrc(geometry param, velocity h_model, seismicData h_seisData)
//{
    //float* wavelet;

    //float f0 = 10.0;                    // source dominawavelet.timeSamplesNt frequency, Hz <]
    //float t0 = 1.2 / f0;                // source padding to move wavelet from left of zero <]

    //float tbytes = h_seisData.timeSamplesNt * sizeof(float);
    //float* time = (float *)malloc(tbytes);
    //wavelet = (float *)malloc(tbytes);

    //// Fill source waveform vector
    //float a = PI * PI * f0 * f0;            // const for wavelet <]
    //float dt2dx2 = (h_seisData.timeStep * h_seisData.timeStep) / (param.modelDx * param.modelDx);   // const for fd stencil <]
    //for (int it = 0; it < h_seisData.timeSamplesNt; it++)
    //{
        //time[it] = it * h_seisData.timeStep;
        //// Ricker wavelet (Mexican hat), second derivative of Gaussian
        //wavelet[it] = 1e10 * (1.0 - 2.0 * a * pow(time[it] - t0, 2)) * exp(-a * pow(time[it] - t0, 2));
        //wavelet[it] *= dt2dx2;
    //}
    //delete[] time;
    ////printf("TIME STEPPING:\n");
    ////printf("\t%e\t:h_seisData.timeStep\n", h_seisData.timeStep);
    ////printf("\t%i\t:h_seisData.timeSamplesNt\n", h_seisData.timeSamplesNt);
    //return wavelet;
//}

//void test_getParameters (geometry param, seismicData h_seisData)
//{
    //cerr<<"param.incShots: "<<param.incShots<<endl;
    //cerr<<"param.incShots: "<<param.incShots<<endl;
    //cerr<<"param.modelDims nx = "<<param.modelNx<<" ny = "<<param.modelNy<<endl;
    //cerr<<"param.modelDx = "<<param.modelDx<<" param.modelDy = "<<param.modelDy<<endl;
    //cerr<<"param.taperBorder = "<<param.taperBorder<<endl;
    //cerr<<"param.nShots "<<param.nShots<<endl;
    //cerr<<"param.nReceptors "<<param.nReceptors<<endl;
    //cerr<<"param.firstReceptorPos "<<param.firstReceptorPos<<endl;
    //cerr<<"param.lastReceptorPos "<<param.lastReceptorPos<<endl;
    //cerr<<"h_seisData.timeSamplesNt "<<h_seisData.timeSamplesNt<<endl;
    //cerr<<"h_seisData.timeStep "<<h_seisData.timeStep<<endl;
//}


/*
===================================================================================
MAIN
===================================================================================
*/
int main(int argc, char *argv[])
{
    /* Main program that reads and writes data and read input variables */
    bool verb;
    sf_init(argc,argv); // init RSF
    if(! sf_getbool("verb",&verb)) verb=0;

    // Setting up I/O files
    sf_file Fvel=NULL;
    Fvel = sf_input("vel");
    sf_file Fshots=NULL;
    Fshots = sf_input("shots");

    // Getting command line parameters
    geometry param = getParameters(Fvel, Fshots);

    // Allocate memory for velocity model
    velocity h_model = getVelFields (Fvel, param);

    cerr<<"vp = "<<h_model.maxVel<<endl;
    cerr<<"param.taperBorder = "<<param.taperBorder<<endl;

    // Taper mask
    float *h_tapermask = tapermask(param);

    // Data
    seismicData h_seisData = allocHostSeisData(param, Fshots);

    // Time stepping
    float* h_wavelet = fillSrc(param, h_model, h_seisData);

    // Set Output files
    int dimensions[3] = {param.modelNy,param.modelNx,1};
    float spacings[3] = {1,1,1};
    int origins[3] = {0,0,0};
    sf_file Fdata = createFile3D("rtm",dimensions,spacings,origins);

    test_getParameters(param, h_seisData);

    // ===================MODELING======================
    rtm(param, h_model, h_wavelet, h_tapermask, h_seisData, Fdata);
    // =================================================

    //printf("Clean memory...");
    delete[] h_model.velField;
    delete[] h_model.extVelField;
    delete[] h_model.firstLayerVelField;
    delete[] h_seisData.seismogram;
    delete[] h_tapermask;

    sf_close();
    return 0;
}
